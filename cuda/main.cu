

#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int print_cuda_version()
{
    int count = 0;

    if (hipSuccess != hipGetDeviceCount(&count)) {
        return -1;
    }

    if (count == 0) {
        return -1;
    }

    for (int device = 0; device < count; ++device) {
        hipDeviceProp_t prop;
        if (hipSuccess == hipGetDeviceProperties(&prop, device)) {
            std::cout << prop.major << "." << prop.minor << std::endl;
        }
    }

    return 0;
}

int main(int argc, char ** argv)
{
    return print_cuda_version();
}

